#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * CUDA Project - HPC 2019 I
 *
 * This program demonstrates a simple simulation box which attempts to model 
 * traffic flow on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector across CUDA threads on the
 * GPU. Only a single thread block is used in this small case, for simplicity.
 * sumArraysOnHost sequentially iterates through vector elements on the host.
 * This version of sumArrays adds host timers to measure GPU and CPU
 * performance.
 */

void checkResult(bool *hostRef, bool *gpuRef, const int N)
{
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (hostRef[i] ^ gpuRef[i])
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %d gpu %d at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}

void show(bool *ip, int size)
{
    // show a bool array
    for (int i = 0; i < size; i++)
    {
        printf("%d ", ip[i]);
    }
    printf("\n");
}

void initialData(bool *ip, int size)
{
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (bool)(rand() & 1);
    }

    return;
}

void simulateFlowOnHost(bool *road_prev, bool *road_curr, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        if (road_prev[idx] == 1 && road_prev[(idx + 1) % N] == 0)
        {
            road_curr[idx] = 0;
            road_curr[(idx + 1) % N] = 1;
        }
    }
}

__global__ void simulateFlowOnGPU(bool *road_prev, bool *road_curr, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N && road_prev[i] == 1 && road_prev[(i + 1) % N] == 0)
    {
        road_curr[i] = 0;
        road_curr[(i + 1) % N] = 1;
    }
}

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    int nElem, nTimes;
    scanf("%d %d", &nElem, &nTimes);
    printf("Vector size %d\n", nElem);
    printf("Number of times %d\n", nTimes);

    // malloc host memory
    size_t nBytes = nElem * sizeof(bool);

    bool *h_road_init, *h_road_curr, *h_road_prev, *hostRef, *gpuRef;
    h_road_init = (bool *)malloc(nBytes);
    h_road_prev = (bool *)malloc(nBytes);
    h_road_curr = (bool *)malloc(nBytes);
    hostRef     = (bool *)malloc(nBytes);
    gpuRef      = (bool *)malloc(nBytes);

    double iStart, iElaps;

    // initialize data at host side
    iStart = seconds();
    initialData(h_road_init, nElem);
    iElaps = seconds() - iStart;
    printf("initialData Time elapsed %f sec\n", iElaps);
    memcpy(h_road_curr, h_road_init, nBytes);
    memset(hostRef, 0, nBytes);
    memset(gpuRef,  0, nBytes);

    // add vector at host side for result checks
    iStart = seconds();
    for (int i = 0; i < nTimes; i++)
    {
        memcpy(h_road_prev, h_road_curr, nBytes);
        simulateFlowOnHost(h_road_prev, h_road_curr, nElem);
    }
    iElaps = seconds() - iStart;
    printf("simulateFlowOnHost Time elapsed %f sec\n", iElaps);
    memcpy(hostRef, h_road_curr, nBytes);

    // malloc device global memory
    bool *d_road_prev, *d_road_curr;
    CHECK(hipMalloc((bool**)&d_road_prev, nBytes));
    CHECK(hipMalloc((bool**)&d_road_curr, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_road_curr, h_road_init, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid  ((nElem + block.x - 1) / block.x);

    iStart = seconds();
    for (int i = 0; i < nTimes; i++)
    {
        CHECK(hipMemcpy(d_road_prev, d_road_curr, nBytes, hipMemcpyDeviceToDevice));
        simulateFlowOnGPU<<<grid, block>>>(d_road_prev, d_road_curr, nElem);
        CHECK(hipDeviceSynchronize());
    }
    iElaps = seconds() - iStart;
    printf("sumArraysOnGPU <<<  %d, %d  >>>  Time elapsed %f sec\n", grid.x,
           block.x, iElaps);

    // check kernel error
    CHECK(hipGetLastError()) ;

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_road_curr, nBytes, hipMemcpyDeviceToHost));
    
    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    CHECK(hipFree(d_road_prev));
    CHECK(hipFree(d_road_curr));

    // free host memory
    free(h_road_init);
    free(h_road_prev);
    free(h_road_curr);
    free(hostRef);
    free(gpuRef);

    return(0);
}